#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////
//Ho Thien Luan -> History Tracking!
// 1. Ver_0: Approximate string matching with k-mismatches
// 2. Ver_1: Optimize by using sharing_memory for storing pattern 
//
//
//
////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>      
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"
#include <time.h>

#define FILENAME_MAXLEN     256
#define THREAD_BLOCK_EXP   (7)
#define THREAD_BLOCK_SIZE  (1 << THREAD_BLOCK_EXP)

__global__ void ASM_kernel(char *g_input_string, int input_size, char *g_pattern, int real_pattern_size, int l_par, int mask, int *g_matrix_M, int *g_matrix_B)
{
    int tid  = threadIdx.x ;
    int gbid = blockIdx.y * gridDim.x + blockIdx.x ;
    int start = gbid*THREAD_BLOCK_SIZE;
    int start_1st_tid = start + tid;
    int start_2nd_tid = start + tid - (real_pattern_size - l_par);
    //unsigned int bit_vector=0;
    int match;
    int sub_sum;
    int sub_init;

    //__shared__ char sub_string_shared [THREAD_BLOCK_SIZE + pattern_length - 1] ;
    __shared__ char pattern_shared [32] ;

//  int pow_2b = 1 << b;
//  unsigned int bit_vector = 0;

//  sub_string_shared[tid] = g_input_string[start+tid];
//  if ( tid < (pattern_length - 1) ){
//     sub_string_shared[THREAD_BLOCK_SIZE + tid] = g_input_string[start+THREAD_BLOCK_SIZE+tid];
//  }
  if ( tid < real_pattern_size ){
     pattern_shared[tid] = g_pattern[tid];
  }
  __syncthreads();


  if(start_1st_tid < real_pattern_size - l_par) {
    //initialization
    sub_sum = 0;
    sub_init = 0;
    for (int i=1; i<=start_1st_tid+1; i++) {
        sub_init = ((sub_init << 1) & mask) | 1;
	sub_sum++;
    }
	//g_matrix_B[start_1st_tid+1] = sub_init;
	//g_matrix_M[start_1st_tid+1] = sub_sum;
    //Processing
    for(int i = 1; i<=real_pattern_size-start_1st_tid; i++) {
        //if (g_input_string[i-1] == g_pattern[start_1st_tid+i]) {match = 0;}
        if (g_input_string[i-1] == pattern_shared[start_1st_tid+i]) {match = 0;}
        else {match = 1;}
        g_matrix_B[(real_pattern_size+1)*(i) + start_1st_tid+i+1] = ((g_matrix_B[(real_pattern_size+1)*(i-1) + (start_1st_tid+i)] << 1) & mask) | match;
        
	sub_sum = 0;
        for (int k = 0; k < l_par; k++) {
           //g_matrix_M[(real_pattern_size+1)*(start_1st_tid+i) + i] += (g_matrix_B[(real_pattern_size+1)*(start_1st_tid+i) + i] >> k) & 1;
           sub_sum += (g_matrix_B[(real_pattern_size+1)*(i) + start_1st_tid+i+1] >> k) & 1;
	   g_matrix_M[(real_pattern_size+1)*(i) + start_1st_tid+i+1] = sub_sum;
        }

    }
	
    
  }

////////////////////////////////////////////////////////////////
  if (start_2nd_tid <= input_size-l_par+1) {
    //initialization
    //g_matrix_B[(real_pattern_size+1)*(start_2nd_tid+0) + 0] = 0;
    //g_matrix_M[(real_pattern_size+1)*(start_2nd_tid+0) + 0] = 0;

    for (int i = 1; i <= real_pattern_size; i++) {
            //if (g_input_string[start_2nd_tid+i-1] == g_pattern[i-1]) {match = 0;}
            if (g_input_string[start_2nd_tid+i-1] == pattern_shared[i-1]) {match = 0;}
            else {match = 1;}
            g_matrix_B[(real_pattern_size+1)*(start_2nd_tid+i) + i] = ((g_matrix_B[(real_pattern_size+1)*(start_2nd_tid+i-1) + (i-1)] << 1) & mask) | match;
	    sub_sum = 0;
            for (int k = 0; k < l_par; k++) {
                //g_matrix_M[(real_pattern_size+1)*(start_2nd_tid+i) + i] += (g_matrix_B[(real_pattern_size+1)*(start_2nd_tid+i) + i] >> k) & 1;
                sub_sum += (g_matrix_B[(real_pattern_size+1)*(start_2nd_tid+i) + i] >> k) & 1;
		g_matrix_M[(real_pattern_size+1)*(start_2nd_tid+i) + i] = sub_sum;
            }
//cuPrintf("threadIdx.x = %d \t ,start = %d, matrix_M = %d, matrix_B = %d, string = %s, pattern = %s, pattern_size = %d\n", tid, start_2nd_tid, g_matrix_M[(real_pattern_size+1)*(start_2nd_tid+i) + i],g_matrix_B[(real_pattern_size+1)*(start_2nd_tid+i) + i], g_input_string[i-1], g_pattern[i-1], l_par);
    }


  }
}

////////////////////////////////
void ASM_process_top (char *g_input_string, size_t input_size,  char *g_pattern, int real_pattern_size, int l_par, int mask, int *g_matrix_M, int *g_matrix_B)
{

    // num_blocks = # of thread blocks to cover input stream
    int num_blocks = (input_size+real_pattern_size-2*l_par+1)/THREAD_BLOCK_SIZE + 1 ;
    //total thread = (m-l) + (n-l+1)


        dim3  dimBlock( THREAD_BLOCK_SIZE, 1 ) ;
        dim3  dimGrid ;

        int p = num_blocks >> 15 ;
        dimGrid.x = num_blocks ;
        if ( p ){
            dimGrid.x = 1<<15 ;
            dimGrid.y = p+1 ;
        }
    cudaPrintfInit();////for cuPrintf

    ASM_kernel <<< dimGrid, dimBlock >>>(g_input_string, input_size, g_pattern, real_pattern_size, l_par, mask, g_matrix_M, g_matrix_B);

    cudaPrintfDisplay();////for cuPrintf
    cudaPrintfEnd();        ////for cuPrintf
}

////////////////////////////////////////////// 
int main(int argc, char **argv)
{
    char inputFile[FILENAME_MAXLEN];
    char patternFile[FILENAME_MAXLEN];
    strcpy( inputFile, argv[2]) ;
    strcpy( patternFile, argv[1]) ;
    int l_par;
    int k_par;
    l_par = strtol(argv[3], NULL, 10);
    k_par = strtol(argv[4], NULL, 10);

////////////////////////////////////////////////////////////////////////////////////
//Process input patterns
    int input_size;
    int pattern_size;
    int real_pattern_size;
    char *h_input_string = NULL ;
    char *h_pattern = NULL ;
    //int  *h_matched_result = NULL ;
    int  *h_matrix_M = NULL ;
    int  *h_matrix_B = NULL ;

    // step 1: read patterns and dump transition table 
//    int deviceID = 0 ;
//    hipDeviceProp_t deviceProp;
//    hipGetDeviceProperties(&deviceProp, deviceID);

    //readPatternFromFile( patternFile) ;
    
    //step 2: prepare input stream
    FILE* fpin = fopen( inputFile, "rb");
    assert ( NULL != fpin ) ;
    
    // obtain file size
    fseek (fpin , 0 , SEEK_END);
    input_size = ftell (fpin);
    rewind (fpin);

    //step2: prepare input pattern 
    FILE* fpattern = fopen( patternFile, "rb");
    assert ( NULL != fpattern ) ;
    
    // obtain file size
    fseek (fpattern , 0 , SEEK_END);
    pattern_size = ftell (fpattern);
    rewind (fpattern);
    
    // allocate memory to contain the whole file
    h_input_string = (char *) malloc (sizeof(char)*input_size);
    assert( NULL != h_input_string );

    h_pattern = (char *) malloc (sizeof(char)*pattern_size);
    assert( NULL != h_pattern );
    real_pattern_size = pattern_size-1; // del char "\n" 

 
    h_matrix_M = (int *) malloc (sizeof(int)*(input_size+1)*(2*pattern_size+1));
    assert( NULL != h_matrix_M );
    memset( h_matrix_M, 0, sizeof(int)*(input_size+1)*(2*pattern_size+1)) ;

    h_matrix_B = (int *) malloc (sizeof(int)*(input_size+1)*(2*pattern_size+1));
    assert( NULL != h_matrix_B );
    memset( h_matrix_B, 0, sizeof(int)*(input_size+1)*(2*pattern_size+1)) ;

    //h_matched_result = (int *) malloc (sizeof(int)*(input_size-l_par+1)*l_par);
    //assert( NULL != h_matched_result );
    //memset( h_matched_result, 0, sizeof(int)*(input_size-l_par+1)*l_par ) ;

    // copy the file into the buffer
    input_size = fread (h_input_string, 1, input_size, fpin);
    fclose(fpin);
    pattern_size = fread (h_pattern, 1, pattern_size, fpattern);
    fclose(fpattern);
     
    //printf("Cir string = %s, length = %d\n", h_pattern, real_pattern_size);
    //Parallel Bit-vector-mismaeches alg.

    #define BIT(x) (1<<(x))
    unsigned int mask = 0;
    for (int i = 0; i < l_par ; i++) {
        mask = (mask << 1) | 1;
    }
/*
    //printf("Mask = %d\n", mask);
    unsigned int match = 0;
    unsigned int bit_vector=0;

    //Bit-vector process
    struct timespec t_start, t_end;
    double elapsedTime;
    clock_gettime (CLOCK_REALTIME, &t_start);
    for (int i = 0; i <= input_size ; i++) {
	h_matrix_B[i*real_pattern_size] = 0;
	h_matrix_M[i*real_pattern_size] = 0;
    }

    for (int i = 1; i <=real_pattern_size; i++) {
	bit_vector = ((bit_vector << 1) & mask) | 1;
	h_matrix_B[i] = bit_vector;	
	for (int j = 0; j < l_par; j++) {
	    h_matrix_M[i] += (bit_vector >> j) & 1;
	}
    //printf("position %d -> h_matrix_B = %u, h_matrix_M = %u\n",i, h_matrix_B[i], h_matrix_M[i]);
    }

    for (int i = 1; i <= real_pattern_size; i++) {
	for (int j = 1; j <= input_size ; j++) {  //circular patterns
	    if (h_input_string[j-1] == h_pattern[i-1]) {match = 0;}
	    else {match = 1;}
	    h_matrix_B[(real_pattern_size+1)*j + i] = ((h_matrix_B[(real_pattern_size+1)*(j-1) + (i-1)] << 1) & mask) | match;
            
	    for (int k = 0; k < l_par; k++) {
            	h_matrix_M[(real_pattern_size+1)*j + i] += (h_matrix_B[(real_pattern_size+1)*j + i] >> k) & 1;
            }
	}	
    }

    clock_gettime(CLOCK_REALTIME, &t_end);
    elapsedTime = (t_end.tv_sec*1000+t_end.tv_nsec/1000000)-(t_start.tv_sec*1000+t_start.tv_nsec/1000000);
*/

    // Process in GPU
    char *g_input_string;
    char *g_pattern;
    int *g_matrix_M;
    int *g_matrix_B;

    hipMalloc (&g_input_string, sizeof(char)*input_size);
    hipMalloc (&g_pattern, sizeof(char)*pattern_size);
    hipMalloc (&g_matrix_M, sizeof(int)*(input_size+1)*(2*pattern_size+1));
    hipMalloc (&g_matrix_B, sizeof(int)*(input_size+1)*(2*pattern_size+1));

    hipMemcpy (g_input_string, h_input_string, sizeof(char)*input_size, hipMemcpyHostToDevice );
    hipMemcpy (g_pattern, h_pattern, sizeof(char)*pattern_size, hipMemcpyHostToDevice );

    // record time setting
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
/*
    unsigned int bit_vector=0;
    for (int i = 0; i <= input_size ; i++) {
        h_matrix_B[i*real_pattern_size] = 0;
        h_matrix_M[i*real_pattern_size] = 0;
    }

    for (int i = 1; i <=real_pattern_size; i++) {
        bit_vector = ((bit_vector << 1) & mask) | 1;
        h_matrix_B[i] = bit_vector;
        for (int j = 0; j < l_par; j++) {
            h_matrix_M[i] += (bit_vector >> j) & 1;
        }
    //printf("position %d -> h_matrix_B = %u, h_matrix_M = %u\n",i, h_matrix_B[i], h_matrix_M[i]);
    }

    hipMemcpy (g_matrix_M, h_matrix_M, sizeof(int)*(input_size+1)*(2*pattern_size+1), hipMemcpyHostToDevice );
    hipMemcpy (g_matrix_B, h_matrix_B, sizeof(int)*(input_size+1)*(2*pattern_size+1), hipMemcpyHostToDevice );
*/
    // step 3: run ASM on GPU           
    ASM_process_top ( g_input_string, input_size, g_pattern, real_pattern_size, l_par, mask, g_matrix_M, g_matrix_B) ;
    //With circular string matching l_par = l;

    // record time setting
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipMemcpy (h_matrix_M, g_matrix_M, sizeof(int)*(input_size+1)*(2*pattern_size+1), hipMemcpyDeviceToHost );
    hipMemcpy (h_matrix_B, g_matrix_B, sizeof(int)*(input_size+1)*(2*pattern_size+1), hipMemcpyDeviceToHost );


    // step 4: output matched result
    int total_result = 0;
    //for (int i = l_par; i<= input_size; i++) {
    for (int i = l_par; i <= input_size; i++) {
        for (int j = l_par; j <= real_pattern_size; j++) {
            //printf("At position %4d, circular pattern %4d : match_M %d, match_B = %d\n", i, j, h_matrix_M[i*(real_pattern_size+1) + j], h_matrix_B[i*(real_pattern_size+1) + j]);
	    if(h_matrix_M[i*(real_pattern_size+1) + j] <= k_par) {total_result++;}
	}
    }
/*
    //Print out Matrix M
    for (int j = 0; j <= real_pattern_size; j++) {
    	for (int i = 0; i<= input_size; i++) {
            printf("%d\t", h_matrix_M[i*(real_pattern_size+1) + j]);
	}
    printf("\n");
    }
*/
    printf("\n\n");
    printf("############################################################\n");
    printf("#--Approximate Circular String Matching with k-Mismatches--#\n");
    printf("#----------------------------------------------------------#\n");
    printf("#---------------Parallel BVM Alg. in GPU-------------------#\n");
    printf("############################################################\n");
    printf("#--Pattern Length            |\t\t %10d \t   #\n",real_pattern_size);
    printf("#----------------------------------------------------------#\n");
    printf("#--Input integer l           |\t\t %10d \t   #\n",l_par);
    printf("#----------------------------------------------------------#\n");
    printf("#--Input Size (bytes)        |\t\t %10d \t   #\n", input_size );
    printf("#----------------------------------------------------------#\n");
    printf("#--Total matched with k = %d  |\t\t %10d \t   #\n", k_par, total_result);
    printf("#----------------------------------------------------------#\n");
    //printf("#--Total elapsed time (ms)   |\t\t %10f \t   #\n", elapsedTime);
    printf("#--Total elapsed time (ms)   |\t\t %10f \t   #\n", time);
    printf("#----------------------------------------------------------#\n");
    //printf("#--Throughput Result (Gbps)  |\t\t %10f \t   #\n", (float)(input_size*8)/(elapsedTime*1000000) );
    //printf("#--Throughput Result (Mbps)  |\t\t %10f \t   #\n", (float)(input_size*8)/(elapsedTime*1000) );
    printf("#--Throughput Result (Gbps)  |\t\t %10f \t   #\n", (float)(input_size*8)/(time*1000000) );
    printf("#--Throughput Result (Mbps)  |\t\t %10f \t   #\n", (float)(input_size*8)/(time*1000) );
    printf("############################################################\n");


    free(h_input_string);
    free(h_pattern);
    //free(h_matched_result); 
    free(h_matrix_M); 
    free(h_matrix_B); 

    hipFree(g_input_string);
    hipFree(g_pattern);
    hipFree(g_matrix_M); 
    hipFree(g_matrix_B); 

            
    return 0;
}
