#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////
//Ho Thien Luan -> History Tracking!
// 1. Ver_0: Approximate string matching with k-mismatches
// 2. Ver_1: Optimize by using sharing_memory for storing pattern 
//
//
//
////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>      
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"
#include <time.h>

#define FILENAME_MAXLEN     256
#define THREAD_BLOCK_EXP   (7)
#define THREAD_BLOCK_SIZE  (1 << THREAD_BLOCK_EXP)

__global__ void ASM_kernel(char *g_input_string, int input_size, int *g_pattern_decode, int real_pattern_size, int mask, int maskplus, int b, int *g_matched_result)
{
    int tid  = threadIdx.x ;
    int gbid = blockIdx.y * gridDim.x + blockIdx.x ;
    int start = gbid*THREAD_BLOCK_SIZE;
    int start_tid = start + tid;
    int pow_2b = 1 << b;
    unsigned long long int bit_vector=0;
    int t_shift;

    __shared__ char sub_string_shared [256] ;
    __shared__ int pattern_decode_shared [4] ;


  sub_string_shared[tid] = g_input_string[start+tid];
  if ( tid < (real_pattern_size - 1) ){
     sub_string_shared[THREAD_BLOCK_SIZE + tid] = g_input_string[start+THREAD_BLOCK_SIZE+tid];
  }
  if ( (real_pattern_size <= tid) && (tid < real_pattern_size + 4) ){
     pattern_decode_shared[tid-real_pattern_size] = g_pattern_decode[tid-real_pattern_size];
  }
  __syncthreads();

////////////////////////////////////////////////////////////////

  if (start_tid < input_size-real_pattern_size+1) {
    for (int i = 0; i < real_pattern_size; i++) {
       t_shift = i%real_pattern_size;  

        if (sub_string_shared[ tid+i ] == 'A') {
            bit_vector = bit_vector + (((pattern_decode_shared[0] << t_shift*b) & mask) | (pattern_decode_shared[0] >> (real_pattern_size - t_shift)*b));
        }
        else if (sub_string_shared[ tid+i ] == 'C'){
            bit_vector = bit_vector + (((pattern_decode_shared[1] << t_shift*b) & mask) | (pattern_decode_shared[1] >> (real_pattern_size - t_shift)*b));
        }
        else if (sub_string_shared[ tid+i ] == 'T'){
            bit_vector = bit_vector + (((pattern_decode_shared[2] << t_shift*b) & mask) | (pattern_decode_shared[2] >> (real_pattern_size - t_shift)*b));
        }
        else if (sub_string_shared[ tid+i ] == 'G'){   //case of G
            bit_vector = bit_vector + (((pattern_decode_shared[3] << t_shift*b) & mask) | (pattern_decode_shared[3] >> (real_pattern_size - t_shift)*b));
        }
        else {  // can be char "\n" 
            bit_vector = bit_vector + maskplus;
        }

    }
        //Get results
           for (int j = 0; j < real_pattern_size ; j++) {  //circular patterns
                //h_matched_result[(i-real_pattern_size+1)*real_pattern_size+(real_pattern_size-1-j)] += ((bit_vector >> (k*real_pattern_size+j)) & 1);
                g_matched_result[start_tid*real_pattern_size+j] = bit_vector % pow_2b;
                bit_vector = bit_vector >> b;
           }

    //cuPrintf("threadIdx.x = %d \t ,start = %d, matrix_M = %d, matrix_B = %d, string = %s, pattern = %s, pattern_size = %d\n", tid, start_tid, g_matrix_M[(real_pattern_size+1)*(start_tid+i) + i],g_matrix_B[(real_pattern_size+1)*(start_tid+i) + i], g_input_string[i-1], g_pattern_circular[i-1], real_pattern_size);
  }

}

////////////////////////////////
void ASM_process_top (char *g_input_string, size_t input_size,  int *g_pattern_decode, int real_pattern_size, int mask, int maskplus, int b, int *g_matched_result)
{

    // num_blocks = # of thread blocks to cover input stream
    int num_blocks = (input_size-real_pattern_size+1)/THREAD_BLOCK_SIZE + 1 ;


        dim3  dimBlock( THREAD_BLOCK_SIZE, 1 ) ;
        dim3  dimGrid ;

        int p = num_blocks >> 15 ;
        dimGrid.x = num_blocks ;
        if ( p ){
            dimGrid.x = 1<<15 ;
            dimGrid.y = p+1 ;
        }
    cudaPrintfInit();////for cuPrintf

    ASM_kernel <<< dimGrid, dimBlock >>>(g_input_string, input_size, g_pattern_decode, real_pattern_size, mask, maskplus, b, g_matched_result);

    cudaPrintfDisplay();////for cuPrintf
    cudaPrintfEnd();        ////for cuPrintf
}


////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    char inputFile[FILENAME_MAXLEN];
    char patternFile[FILENAME_MAXLEN];
    strcpy( inputFile, argv[2]) ;
    strcpy( patternFile, argv[1]) ;
    int k_par;
    k_par = strtol(argv[3], NULL, 10);

////////////////////////////////////////////////////////////////////////////////////
//Process input patterns
    int input_size;
    int pattern_size;
    int real_pattern_size;
    char *h_input_string = NULL ;
    char *h_pattern = NULL ;
    int  *h_matched_result = NULL ;
    int  *h_pattern_decode = (int*) malloc( sizeof(int)*4 ) ;

    // step 1: read patterns and dump transition table 
//    int deviceID = 0 ;
//    hipDeviceProp_t deviceProp;
//    hipGetDeviceProperties(&deviceProp, deviceID);

    //readPatternFromFile( patternFile) ;
    
    //step 2: prepare input stream
    FILE* fpin = fopen( inputFile, "rb");
    assert ( NULL != fpin ) ;
    
    // obtain file size
    fseek (fpin , 0 , SEEK_END);
    input_size = ftell (fpin);
    rewind (fpin);

    //step2: prepare input pattern 
    FILE* fpattern = fopen( patternFile, "rb");
    assert ( NULL != fpattern ) ;
    
    // obtain file size
    fseek (fpattern , 0 , SEEK_END);
    pattern_size = ftell (fpattern);
    rewind (fpattern);
    
    // allocate memory to contain the whole file
    h_input_string = (char *) malloc (sizeof(char)*input_size);
    assert( NULL != h_input_string );

    h_pattern = (char *) malloc (sizeof(char)*pattern_size);
    assert( NULL != h_pattern );
    real_pattern_size = pattern_size-1;

 
    h_matched_result = (int *) malloc (sizeof(int)*(input_size-real_pattern_size+1)*real_pattern_size);
    assert( NULL != h_matched_result );
    memset( h_matched_result, 0, sizeof(int)*(input_size-real_pattern_size+1)*real_pattern_size) ;

    // copy the file into the buffer
    input_size = fread (h_input_string, 1, input_size, fpin);
    fclose(fpin);
    pattern_size = fread (h_pattern, 1, pattern_size, fpattern);
    fclose(fpattern);
     
    //printf("Cir string = %s, length = %d\n", h_pattern, real_pattern_size);

    //ACSM Preprocess - Define table T[]
    unsigned long long int T_A = 0;
	int cal_A = 0;
    unsigned long long int T_C = 0;
	int cal_C = 0;
    unsigned long long int T_T = 0;
	int cal_T = 0;
    unsigned long long int T_G = 0;
	int cal_G = 0;

float sub_real_pattern_size = real_pattern_size+1;
float float_b = log2(sub_real_pattern_size);
int b = float_b;
if(b != float_b) {
   b=b+1;
}
//int b = log2(sub_real_pattern_size)+1;
//int pow_2b = 1 << b;
//printf("#-pattern = %d, b=%d, float_b = %f, opw_2b = %d-#\n",real_pattern_size,b,float_b,pow_2b);

    //for ( int i = real_pattern_size-1; i>=0; i--) {
    for ( int i = 0; h_pattern[ i ]; i++) {
      if(h_pattern[ i ] != '\n') {
        //printf("Process for char: %c\n", h_pattern[ i ]);
	if (h_pattern[ i ] == 'A') {
	    cal_A = 0;
	    cal_C = 1;
	    cal_T = 1;
	    cal_G = 1;
	}
	else if (h_pattern[ i ] == 'C'){
	    cal_A = 1;
	    cal_C = 0;
	    cal_T = 1;
	    cal_G = 1;
	}
	else if (h_pattern[ i ] == 'T'){
	    cal_A = 1;
	    cal_C = 1;
	    cal_T = 0;
	    cal_G = 1;
	}
	else if (h_pattern[ i ] == 'G'){
	    cal_A = 1;
	    cal_C = 1;
	    cal_T = 1;
	    cal_G = 0;
	}

	T_A = (T_A << b) + cal_A;
	T_C = (T_C << b) + cal_C;
	T_T = (T_T << b) + cal_T;
	T_G = (T_G << b) + cal_G;

      }
    }
	h_pattern_decode[0] = T_A;
	h_pattern_decode[1] = T_C;
	h_pattern_decode[2] = T_T;
	h_pattern_decode[3] = T_G;
     
    //printf("\nT_A: %d\n", T_A);
    //printf("\nT_C: %d\n", T_C);
    //printf("\nT_T: %d\n", T_T);
    //printf("\nT_G: %d\n", T_G);

    //shift-add bit-vector.
    //unsigned long long int bit_vector=0;
    //int t_shift = 0;
    #define BIT(x) (1<<(x))
    unsigned long long int mask = 0;
    for (int i = 0; i < real_pattern_size*b ; i++) {
	mask = (mask << 1) | 1;
    }
    unsigned long long int maskplus = 0;
    for (int i = 0; i < real_pattern_size ; i++) {
	maskplus = (maskplus << b) | 1;
    }

/*
    //ACSM process
    struct timespec t_start, t_end;
    double elapsedTime;
    clock_gettime (CLOCK_REALTIME, &t_start);

    //for (int i = 0; h_input_string [ i ] ; i++) {
    for (int i = 0; i<input_size-(real_pattern_size-1) ; i++) {
	bit_vector = 0 ;
      for(int k=0; k< real_pattern_size; k++) {

	t_shift = k%real_pattern_size;
        //printf("Process for char: %c, T_A = %u, t_shift = %d\n", h_input_string[ i ], T_A, t_shift);
	

	if (h_input_string[ i+k ] == 'A') {
	    bit_vector = bit_vector + ((T_A >> t_shift*b) & mask);
	}
	else if (h_input_string[ i+k ] == 'C'){
	    bit_vector = bit_vector + ((T_C >> t_shift*b) & mask);
	}
	else if (h_input_string[ i+k ] == 'T'){
	    bit_vector = bit_vector + ((T_T >> t_shift*b) & mask);
	}
	else if (h_input_string[ i+k ] == 'G'){   //case of G
	    bit_vector = bit_vector + ((T_G >> t_shift*b) & mask);
	}
	else {  // can be char "\n" 
	    bit_vector = bit_vector + maskplus; 
	}
	//Get results
//	for (int j = no_of_patterns-1; j >= 0; j--) {
//             h_matched_result[i*no_of_patterns+j] = vector % pow_2b;
//             vector = vector >> b;
//        }

    	//printf("bit_vector: %u\n", bit_vector);
		
      }
	   for (int j = 0; j < real_pattern_size ; j++) {  //circular patterns
		//h_matched_result[(i-real_pattern_size+1)*real_pattern_size+(real_pattern_size-1-j)] += ((bit_vector >> (k*real_pattern_size+j)) & 1);
                h_matched_result[i*real_pattern_size+j] = bit_vector % pow_2b;
                bit_vector = bit_vector >> b;
	   }
    }// for h_input_string

    clock_gettime(CLOCK_REALTIME, &t_end);
    elapsedTime = (t_end.tv_sec*1000+t_end.tv_nsec/1000000)-(t_start.tv_sec*1000+t_start.tv_nsec/1000000);
*/

    //Process in GPU
    char *g_input_string;
    int *g_matched_result;
    int *g_pattern_decode;

    hipMalloc (&g_input_string, sizeof(char)*input_size);
    hipMalloc (&g_matched_result, sizeof(int)*(input_size-real_pattern_size+1)*real_pattern_size);
    hipMalloc (&g_pattern_decode, sizeof(int)*4);

    hipMemcpy (g_input_string, h_input_string, sizeof(char)*input_size, hipMemcpyHostToDevice );
    hipMemcpy (g_pattern_decode, h_pattern_decode, sizeof(int)*4, hipMemcpyHostToDevice);


    // record time setting
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    // step 3: run ASM on GPU           
    ASM_process_top ( g_input_string, input_size, g_pattern_decode, real_pattern_size, mask, maskplus, b, g_matched_result) ;

    // record time setting
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipMemcpy (h_matched_result, g_matched_result, sizeof(int)*(input_size-real_pattern_size+1)*real_pattern_size, hipMemcpyDeviceToHost );


    // step 4: output matched result
    int total_result = 0;
    for (int i = 0; i < input_size-(real_pattern_size-1); i++) {
        for (int j = 0; j < real_pattern_size; j++) {
            //printf("At position %4d, circular pattern %4d : match pattern %d\n", i, j, h_matched_result[i*real_pattern_size + j]);
	    if(h_matched_result[i*real_pattern_size + j] <= k_par) {total_result++;}
	}
    }
    printf("\n\n");
    printf("############################################################\n");
    printf("#--Approximate Circular String Matching with k-Mismatches--#\n");
    printf("#----------------------------------------------------------#\n");
    printf("#----------Modified shared-mem PCVM Alg. in GPU------------#\n");
    printf("############################################################\n");
    printf("#--Pattern Length            |\t\t %10d \t   #\n",real_pattern_size);
    printf("#----------------------------------------------------------#\n");
    printf("#--Input Size (bytes)        |\t\t %10d \t   #\n", input_size );
    printf("#----------------------------------------------------------#\n");
    printf("#--Total matched with k = %d  |\t\t %10d \t   #\n", k_par, total_result);
    printf("#----------------------------------------------------------#\n");
    //printf("#--Total elapsed time (ms)   |\t\t %10f \t   #\n", elapsedTime);
    printf("#--Total elapsed time (ms)   |\t\t %10f \t   #\n", time);
    printf("#----------------------------------------------------------#\n");
    //printf("#--Throughput Result (Gbps)  |\t\t %10f \t   #\n", (float)(input_size*8)/(elapsedTime*1000000) );
    //printf("#--Throughput Result (Mbps)  |\t\t %10f \t   #\n", (float)(input_size*8)/(elapsedTime*1000) );
    printf("#--Throughput Result (Gbps)  |\t\t %10f \t   #\n", (float)(input_size*8)/(time*1000000) );
    printf("#--Throughput Result (Mbps)  |\t\t %10f \t   #\n", (float)(input_size*8)/(time*1000) );
    printf("############################################################\n");


    free(h_input_string);
    free(h_pattern);
    free(h_matched_result); 

    hipFree(g_input_string);
    hipFree(g_pattern_decode);
    hipFree(g_matched_result);
            
    return 0;
}
