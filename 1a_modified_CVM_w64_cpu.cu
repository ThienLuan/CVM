#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////
//Ho Thien Luan -> History Tracking!
// 1. Ver_0: Approximate string matching with k-mismatches
// 2. Ver_1: Optimize by using sharing_memory for storing pattern 
//
//
//
////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>      
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"
#include <time.h>

#define FILENAME_MAXLEN     256
////////////////////////////////////////////////////////////////////////////////////
 
int main(int argc, char **argv)
{
    char inputFile[FILENAME_MAXLEN];
    char patternFile[FILENAME_MAXLEN];
    strcpy( inputFile, argv[2]) ;
    strcpy( patternFile, argv[1]) ;
    int k_par;
    k_par = strtol(argv[3], NULL, 10);

////////////////////////////////////////////////////////////////////////////////////
//Process input patterns
    int input_size;
    int pattern_size;
    int real_pattern_size;
    char *h_input_string = NULL ;
    char *h_pattern = NULL ;
    int  *h_matched_result = NULL ;

    // step 1: read patterns and dump transition table 
//    int deviceID = 0 ;
//    hipDeviceProp_t deviceProp;
//    hipGetDeviceProperties(&deviceProp, deviceID);

    //readPatternFromFile( patternFile) ;
    
    //step 2: prepare input stream
    FILE* fpin = fopen( inputFile, "rb");
    assert ( NULL != fpin ) ;
    
    // obtain file size
    fseek (fpin , 0 , SEEK_END);
    input_size = ftell (fpin);
    rewind (fpin);

    //step2: prepare input pattern 
    FILE* fpattern = fopen( patternFile, "rb");
    assert ( NULL != fpattern ) ;
    
    // obtain file size
    fseek (fpattern , 0 , SEEK_END);
    pattern_size = ftell (fpattern);
    rewind (fpattern);
    
    // allocate memory to contain the whole file
    h_input_string = (char *) malloc (sizeof(char)*input_size);
    assert( NULL != h_input_string );

    h_pattern = (char *) malloc (sizeof(char)*pattern_size);
    assert( NULL != h_pattern );
    real_pattern_size = pattern_size-1;
 
    h_matched_result = (int *) malloc (sizeof(int)*(input_size-real_pattern_size+1)*real_pattern_size);
    assert( NULL != h_matched_result );
    memset( h_matched_result, 0, sizeof(int)*(input_size-real_pattern_size+1)*real_pattern_size) ;

    // copy the file into the buffer
    input_size = fread (h_input_string, 1, input_size, fpin);
    fclose(fpin);
    pattern_size = fread (h_pattern, 1, pattern_size, fpattern);
    fclose(fpattern);
     
    //printf("Cir string = %s, length = %d\n", h_pattern, real_pattern_size);

    //ACSM Preprocess - Define table T[]
    unsigned long long int T_A = 0;
	int cal_A = 0;
    unsigned long long int T_C = 0;
	int cal_C = 0;
    unsigned long long int T_T = 0;
	int cal_T = 0;
    unsigned long long int T_G = 0;
	int cal_G = 0;

float sub_real_pattern_size = real_pattern_size+1;
float float_b = log2(sub_real_pattern_size);
int b = float_b;
if(b != float_b) {
   b=b+1;
}
//int b = log2(sub_real_pattern)+1;
int pow_2b = 1 << b;
//printf("#-pattern = %d, b=%d, float_b = %f, opw_2b = %d-#\n",real_pattern_size,b,float_b,pow_2b);

    //for ( int i = real_pattern_size-1; i>=0; i--) {
    for ( int i = 0; h_pattern[ i ]; i++) {
      if(h_pattern[ i ] != '\n') {
        //printf("Process for char: %c\n", h_pattern[ i ]);
	if (h_pattern[ i ] == 'A') {
	    cal_A = 0;
	    cal_C = 1;
	    cal_T = 1;
	    cal_G = 1;
	}
	else if (h_pattern[ i ] == 'C'){
	    cal_A = 1;
	    cal_C = 0;
	    cal_T = 1;
	    cal_G = 1;
	}
	else if (h_pattern[ i ] == 'T'){
	    cal_A = 1;
	    cal_C = 1;
	    cal_T = 0;
	    cal_G = 1;
	}
	else if (h_pattern[ i ] == 'G'){
	    cal_A = 1;
	    cal_C = 1;
	    cal_T = 1;
	    cal_G = 0;
	}

	T_A = (T_A << b) + cal_A;
	T_C = (T_C << b) + cal_C;
	T_T = (T_T << b) + cal_T;
	T_G = (T_G << b) + cal_G;

      }
    }
     
    //printf("\nT_A: %d\n", T_A);
    //printf("\nT_C: %d\n", T_C);
    //printf("\nT_T: %d\n", T_T);
    //printf("\nT_G: %d\n", T_G);

    //shift-add bit-vector.
    unsigned long long int bit_vector=0;
    int t_shift = 0;
    //int position = 0;
    #define BIT(x) (1<<(x))
    unsigned long long int mask = 0;
    for (int i = 0; i < real_pattern_size*b ; i++) {
	mask = (mask << 1) | 1;
    }
    unsigned long long int maskplus = 0;
    for (int i = 0; i < real_pattern_size ; i++) {
	maskplus = (maskplus << b) | 1;
    }

    //ACSM process
    struct timespec t_start, t_end;
    double elapsedTime;
    clock_gettime (CLOCK_REALTIME, &t_start);

    //for (int i = 0; h_input_string [ i ] ; i++) {
    for (int i = 0; i<input_size-(real_pattern_size-1) ; i++) {
	bit_vector = 0 ;
      for(int k=0; k<real_pattern_size; k++) {

	t_shift = k%real_pattern_size;
        //printf("Process for char: %c, T_A = %u, t_shift = %d\n", h_input_string[ i ], T_A, t_shift);
	

	if (h_input_string[ i+k ] == 'A') {
	    bit_vector = bit_vector + (((T_A << t_shift*b) & mask) | (T_A >> (real_pattern_size - t_shift)*b));
	}
	else if (h_input_string[ i+k ] == 'C'){
	    bit_vector = bit_vector + (((T_C << t_shift*b) & mask) | (T_C >> (real_pattern_size - t_shift)*b));
	}
	else if (h_input_string[ i+k ] == 'T'){
	    bit_vector = bit_vector + (((T_T << t_shift*b) & mask) | (T_T >> (real_pattern_size - t_shift)*b));
	}
	else if (h_input_string[ i+k ] == 'G'){   //case of G
	    bit_vector = bit_vector + (((T_G << t_shift*b) & mask) | (T_G >> (real_pattern_size - t_shift)*b));
	}
	else {  // can be char "\n" 
	    bit_vector = bit_vector + maskplus; 
	}
	//Get results
//	for (int j = no_of_patterns-1; j >= 0; j--) {
//             h_matched_result[i*no_of_patterns+j] = vector % pow_2b;
//             vector = vector >> b;
//        }

    	//printf("bit_vector: %u\n", bit_vector);
		
      }
	   for (int j = 0; j < real_pattern_size; j++) {  //circular patterns
		//h_matched_result[(i-real_pattern_size+1)*real_pattern_size+(real_pattern_size-1-j)] += ((bit_vector >> (k*real_pattern_size+j)) & 1);
                h_matched_result[i*real_pattern_size+j] = bit_vector % pow_2b;
                bit_vector = bit_vector >> b;
	   }
    }// for h_input_string

    clock_gettime(CLOCK_REALTIME, &t_end);
    elapsedTime = (t_end.tv_sec*1000+t_end.tv_nsec/1000000)-(t_start.tv_sec*1000+t_start.tv_nsec/1000000);
 
    // step 4: output matched result
    int total_result = 0;
    for (int i = 0; i < input_size-(real_pattern_size-1); i++) {
        for (int j = 0; j < real_pattern_size; j++) {
            //printf("At position %4d, circular pattern %4d : match pattern %d\n", i, j, h_matched_result[i*real_pattern_size + j]);
	    if(h_matched_result[i*real_pattern_size + j] <= k_par) {total_result++;}
	}
    }
    printf("\n\n");
    printf("############################################################\n");
    printf("#--Approximate Circular String Matching with k-Mismatches--#\n");
    printf("#----------------------------------------------------------#\n");
    printf("#--------------Modified CVM Alg. in CPU--------------------#\n");
    printf("############################################################\n");
    printf("#--Pattern Length            |\t\t %10d \t   #\n", real_pattern_size);
    printf("#----------------------------------------------------------#\n");
    printf("#--Input Size (bytes)        |\t\t %10d \t   #\n", input_size );
    printf("#----------------------------------------------------------#\n");
    printf("#--Total matched with k = %d  |\t\t %10d \t   #\n", k_par, total_result);
    printf("#----------------------------------------------------------#\n");
    printf("#--Total elapsed time (ms)   |\t\t %10f \t   #\n", elapsedTime);
    printf("#----------------------------------------------------------#\n");
    printf("#--Throughput Result (Gbps)  |\t\t %10f \t   #\n", (float)(input_size*8)/(elapsedTime*1000000) );
    printf("#--Throughput Result (Mbps)  |\t\t %10f \t   #\n", (float)(input_size*8)/(elapsedTime*1000) );
    printf("############################################################\n");


    free(h_input_string);
    free(h_pattern);
    free(h_matched_result); 
            
    return 0;
}
