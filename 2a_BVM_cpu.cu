////////////////////////////////////////////////////////////
//Ho Thien Luan -> History Tracking!
// 1. Ver_0: Approximate string matching with k-mismatches
// 2. Ver_1: Optimize by using sharing_memory for storing pattern 
//
//
//
////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>      
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"
#include <time.h>

#define FILENAME_MAXLEN     256
 
int main(int argc, char **argv)
{
    char inputFile[FILENAME_MAXLEN];
    char patternFile[FILENAME_MAXLEN];
    strcpy( inputFile, argv[2]) ;
    strcpy( patternFile, argv[1]) ;
    int l_par;
    int k_par;
    l_par = strtol(argv[3], NULL, 10);
    k_par = strtol(argv[4], NULL, 10);

////////////////////////////////////////////////////////////////////////////////////
//Process input patterns
    int input_size;
    int pattern_size;
    int real_pattern_size;
    char *h_input_string = NULL ;
    char *h_pattern = NULL ;
    //int  *h_matched_result = NULL ;
    int  *h_matrix_M = NULL ;
    int  *h_matrix_B = NULL ;

    // step 1: read patterns and dump transition table 
//    int deviceID = 0 ;
//    hipDeviceProp_t deviceProp;
//    hipGetDeviceProperties(&deviceProp, deviceID);

    //readPatternFromFile( patternFile) ;
    
    //step 2: prepare input stream
    FILE* fpin = fopen( inputFile, "rb");
    assert ( NULL != fpin ) ;
    
    // obtain file size
    fseek (fpin , 0 , SEEK_END);
    input_size = ftell (fpin);
    rewind (fpin);

    //step2: prepare input pattern 
    FILE* fpattern = fopen( patternFile, "rb");
    assert ( NULL != fpattern ) ;
    
    // obtain file size
    fseek (fpattern , 0 , SEEK_END);
    pattern_size = ftell (fpattern);
    rewind (fpattern);
    
    // allocate memory to contain the whole file
    h_input_string = (char *) malloc (sizeof(char)*input_size);
    assert( NULL != h_input_string );

    h_pattern = (char *) malloc (sizeof(char)*pattern_size);
    assert( NULL != h_pattern );
    real_pattern_size = pattern_size-1; // del char "\n" 

 
    h_matrix_M = (int *) malloc (sizeof(int)*(input_size+1)*(2*pattern_size+1));
    assert( NULL != h_matrix_M );
    memset( h_matrix_M, 0, sizeof(int)*(input_size+1)*(2*pattern_size+1)) ;

    h_matrix_B = (int *) malloc (sizeof(int)*(input_size+1)*(2*pattern_size+1));
    assert( NULL != h_matrix_B );
    memset( h_matrix_B, 0, sizeof(int)*(input_size+1)*(2*pattern_size+1)) ;

    //h_matched_result = (int *) malloc (sizeof(int)*(input_size-l_par+1)*l_par);
    //assert( NULL != h_matched_result );
    //memset( h_matched_result, 0, sizeof(int)*(input_size-l_par+1)*l_par ) ;

    // copy the file into the buffer
    input_size = fread (h_input_string, 1, input_size, fpin);
    fclose(fpin);
    pattern_size = fread (h_pattern, 1, pattern_size, fpattern);
    fclose(fpattern);
     
    //printf("Cir string = %s, length = %d\n", h_pattern, real_pattern_size);

    //Bit-vector-mismatches Alg.
    unsigned int bit_vector=0;
    //int position = 0;
    #define BIT(x) (1<<(x))
    unsigned int mask = 0;
    for (int i = 0; i < l_par ; i++) {
        mask = (mask << 1) | 1;
    }
    //printf("Mask = %d\n", mask);
    unsigned int match = 0;

    //Bit-vector process
    struct timespec t_start, t_end;
    double elapsedTime;
    clock_gettime (CLOCK_REALTIME, &t_start);
    for (int i = 0; i <= input_size ; i++) {
	h_matrix_B[i*real_pattern_size] = 0;
	h_matrix_M[i*real_pattern_size] = 0;
    }

    for (int i = 1; i <=real_pattern_size; i++) {
	bit_vector = ((bit_vector << 1) & mask) | 1;
	h_matrix_B[i] = bit_vector;	
	for (int j = 0; j < l_par; j++) {
	    h_matrix_M[i] += (bit_vector >> j) & 1;
	}
    //printf("position %d -> h_matrix_B = %u, h_matrix_M = %u\n",i, h_matrix_B[i], h_matrix_M[i]);
    }

    for (int i = 1; i <= real_pattern_size; i++) {
	for (int j = 1; j <= input_size ; j++) {  //circular patterns
	    if (h_input_string[j-1] == h_pattern[i-1]) {match = 0;}
	    else {match = 1;}
	    h_matrix_B[(real_pattern_size+1)*j + i] = ((h_matrix_B[(real_pattern_size+1)*(j-1) + (i-1)] << 1) & mask) | match;
            
	    for (int k = 0; k < l_par; k++) {
            	h_matrix_M[(real_pattern_size+1)*j + i] += (h_matrix_B[(real_pattern_size+1)*j + i] >> k) & 1;
            }
	}	
    }

    clock_gettime(CLOCK_REALTIME, &t_end);
    elapsedTime = (t_end.tv_sec*1000+t_end.tv_nsec/1000000)-(t_start.tv_sec*1000+t_start.tv_nsec/1000000);

 
    // step 4: output matched result
    int total_result = 0;
    //for (int i = l_par; i<= input_size; i++) {
    for (int i = l_par; i <= input_size; i++) {
        for (int j = l_par; j <= real_pattern_size; j++) {
            //printf("At position %4d, circular pattern %4d : match pattern %d\n", i, j,  h_matrix_M[i*(real_pattern_size+1) + j]);
	    if(h_matrix_M[i*(real_pattern_size+1) + j] <= k_par) {total_result++;}
	}
    }
/*
    //Print out Matrix M
    for (int j = 0; j <= real_pattern_size; j++) {
    	for (int i = 0; i<= input_size; i++) {
            printf("%d\t", h_matrix_M[i*(real_pattern_size+1) + j]);
	}
    printf("\n");
    }
*/
    printf("\n\n");
    printf("############################################################\n");
    printf("#--Approximate Circular String Matching with k-Mismatches--#\n");
    printf("#----------------------------------------------------------#\n");
    printf("#-----------Bit-vector-mismatches Alg. in CPU--------------#\n");
    printf("############################################################\n");
    printf("#--Pattern Length            |\t\t %10d \t   #\n",real_pattern_size);
    printf("#----------------------------------------------------------#\n");
    printf("#--Input integer l           |\t\t %10d \t   #\n",l_par);
    printf("#----------------------------------------------------------#\n");
    printf("#--Input Size (bytes)        |\t\t %10d \t   #\n", input_size );
    printf("#----------------------------------------------------------#\n");
    printf("#--Total matched with k = %d  |\t\t %10d \t   #\n", k_par, total_result);
    printf("#----------------------------------------------------------#\n");
    printf("#--Total elapsed time (ms)   |\t\t %10f \t   #\n", elapsedTime);
    printf("#----------------------------------------------------------#\n");
    printf("#--Throughput Result (Gbps)  |\t\t %10f \t   #\n", (float)(input_size*8)/(elapsedTime*1000000) );
    printf("#--Throughput Result (Mbps)  |\t\t %10f \t   #\n", (float)(input_size*8)/(elapsedTime*1000) );
    printf("############################################################\n");


    free(h_input_string);
    free(h_pattern);
    //free(h_matched_result); 
    free(h_matrix_M); 
    free(h_matrix_B); 

            
    return 0;
}
